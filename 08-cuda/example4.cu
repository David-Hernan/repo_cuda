#include "hip/hip_runtime.h"
// =================================================================
//
// File: example4.cu
// Author(s):
// Description: This file contains the code to count the number of
//				even numbers within an array using CUDA.
//
// =================================================================

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "utils.h"

#define SIZE 1000000000 //1e9
#define THREADS	256
#define BLOCKS	MMIN(32, ((SIZE / THREADS) + 1))
// implement your code
__global__ void even(int *array, int *results) {
	__shared__ int cache[THREADS];

	int tid = threadIdx.x + (blockIdx.x * blockDim.x);
	int cacheIndex = threadIdx.x;

	int result = 0;
	while (tid < SIZE) {
		//aux = (aux < array[tid])? aux : array[tid];
    if((array[tid]%2)==0){
      result ++;
    }
		tid += blockDim.x * gridDim.x;
	}

	cache[cacheIndex] = result;

	__syncthreads();

	int i = blockDim.x / 2;
	while (i > 0) {
		if (cacheIndex < i) {
			cache[cacheIndex] = even(cache[cacheIndex], cache[cacheIndex + 1]);
		}
		__syncthreads();
		i /= 2;
	}

	if (cacheIndex == 0) {
		results[blockIdx.x] = cache[cacheIndex];
	}
}

int main(int argc, char* argv[]) {
	int i, *a, *results;
  int *d_a, *d_r;
	double ms;

	a = (int *) malloc(sizeof(int) * SIZE);
	random_array(a, SIZE);
	display_array("a", a);

  results = (int *) malloc(sizeof(int) * BLOCKS);

	hipMalloc( (void**) &d_a, SIZE * sizeof(int) );
	hipMalloc( (void**) &d_r, BLOCKS * sizeof(int) );

	hipMemcpy(d_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);

	printf("Starting...\n");
	ms = 0;
	for (i = 1; i <= N; i++) {
		start_timer();

		even<<<BLOCKS, THREADS>>> (d_a, d_r);

		ms += stop_timer();
	}

	hipMemcpy(results, d_r, BLOCKS * sizeof(int), hipMemcpyDeviceToHost);

	int aux = INT_MAX;
	for (i = 0; i < BLOCKS; i++) {
		aux = MMIN(aux, results[i]);
	}

	printf("result = %i\n", aux);
	printf("avg time = %.5lf\n", (ms / N));

	hipFree(d_r);
	hipFree(d_a);

	free(a);
  free(results);
	return 0;
}
