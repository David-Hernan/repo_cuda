#include "hip/hip_runtime.h"
/*----------------------------------------------------------------
*
* Multiprocesadores: CUDA
* Fecha: 14-Nov-2021
* Autores:
* A01173130 David Hernán García Fernández
* A01701434 Joseph Alessandro García García
*
*--------------------------------------------------------------*/

// =================================================================
//
// File: example8.cu
// Author(s):
// Description: This file contains the code that implements the
//				enumeration sort algorithm using CUDA.
//
// Copyright (c) 2020 by Tecnologico de Monterrey.
// All Rights Reserved. May be reproduced for any non-commercial
// purpose.
//
// =================================================================
//Tiempo de ejecución paralelo:        ms
//Tiempo de ejecución secuencial:  ms
//Speed Up:

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "utils.h"

#define SIZE 10000
#define THREADS 256
#define BLOCKS	MMIN(32, ((SIZE / THREADS) + 1))

__global__ void even(int* arr, int size) {
  int i, aux;

  i = (threadIdx.x * 2);
  if (i <= size - 2) {
    if (arr[i] > arr[i + 1]) {
      aux = arr[i];
      arr[i] = arr[i + 1];
      arr[i + 1] = aux;
    }
  }
}

__global__ void odd(int* arr, int size) {
  int i, aux;

  i = (threadIdx.x * 2) + 1;
  if (i <= size - 2) {
    if (arr[i] > arr[i + 1]) {
      aux = arr[i];
      arr[i] = arr[i + 1];
      arr[i + 1] = aux;
    }
  }
}

int main(int argc, char* argv[]) {
	int i, j, *a, *d_a;
	double ms;

	a = (int *) malloc(sizeof(int) * SIZE);
	random_array(a, SIZE);
	display_array("a", a);

	hipMalloc( (void**) &d_a, SIZE * sizeof(int) );

  printf("Starting...\n");
	ms = 0;
	for (i = 1; i <= N; i++) {
    hipMemcpy(d_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);

		start_timer();

    for (j = 0; j <= SIZE / 2; j++) {
      even<<<1, THREADS>>>(d_a, SIZE);
      odd<<<1, THREADS>>>(d_a,SIZE);
    }

		ms += stop_timer();
	}

  hipMemcpy(a, d_a, SIZE * sizeof(int), hipMemcpyDeviceToHost);
	display_array("a", a);
  printf("avg time = %.5lf ms\n", (ms / N));

	hipFree(d_a);

	free(a);

  return 0;
}
