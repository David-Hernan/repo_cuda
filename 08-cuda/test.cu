#include "hip/hip_runtime.h"
// =================================================================
//
// File: example7.cu
// Author(s):
// Description: This file contains the code to brute-force all
//				prime numbers less than MAXIMUM using CUDA.
//
// Copyright (c) 2020 by Tecnologico de Monterrey.
// All Rights Reserved. May be reproduced for any non-commercial
// purpose.
//
// =================================================================

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "utils.h"

#define MAXIMUM 10000 //1e6 1000000
#define THREADS 256
#define BLOCKS	MMIN(32, ((MAXIMUM / THREADS) + 1))

__global__ void GetPrime(int *array, int pos) {

	int tid = threadIdx.x + (blockIdx.x * blockDim.x);

	int aux, b, i;
	while (tid < MAXIMUM) {
        // array[tid]=1;
        // aux=1;
        // for(i = 2; i <= sqrt((float)tid); i++){
        //         if(!(tid%i)){
        //             aux=0;
        //         }
		// 	}
        //     array[tid]=aux;
        //     tid += blockDim.x * gridDim.x;
        // }

			b=i/2;
			if(b<2){
				b=2;
			}
			aux=0;
			for(i = 2; i <= b; i++){
				if(array[tid-1] % i == 0){
					aux=0;
				}else{
					aux=1;
				}
			}
			array[tid]=aux;
            tid += blockDim.x * gridDim.x;
	}
}

int main(int argc, char* argv[]) {
	int i, *a, *d_a;
	double ms;

	a = (int*) malloc(sizeof(int) * MAXIMUM);
	fill_array(a, MAXIMUM);
	display_array("before", a);

    hipMalloc( (void**) &d_a, MAXIMUM * sizeof(int) );
    hipMemcpy(d_a, a, MAXIMUM * sizeof(int), hipMemcpyHostToDevice);

    printf("\nStarting...\n");
	ms = 0;
	for (i = 1; i <= N; i++) {
		start_timer();

        GetPrime<<<BLOCKS, THREADS>>> (d_a, MAXIMUM);

		ms += stop_timer();
	}

    hipMemcpy(a, d_a, MAXIMUM * sizeof(int), hipMemcpyDeviceToHost);


	for (i = 2; i < TOP_VALUE; i++) {
		if(a[i] == 1){
            printf("%i ",i);
        }
	}

    printf("\navg time = %.5lf ms\n", (ms / N));

    hipFree(d_a);
	free(a);

    return 0;
}
