#include "hip/hip_runtime.h"
// =================================================================
//
// File: example7.cu
// Author(s):
// Description: This file contains the code to brute-force all
//				prime numbers less than MAXIMUM using CUDA.
//
// Copyright (c) 2020 by Tecnologico de Monterrey.
// All Rights Reserved. May be reproduced for any non-commercial
// purpose.
//
// =================================================================

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "utils.h"

#define MAXIMUM 10000 //1e6 1000000
#define THREADS 256
//#define BLOCKS	MMIN(32, ((SIZE / THREADS) + 1))
#define BLOCKS	MMIN(32, ((MAXIMUM / THREADS) + 1))
// implement your code
__global__ void even(int* arr, int size) {
  int tid = threadIdx.x + (blockIdx.x * blockDim.x);

  int j, prime;

  while (tid < MAXIMUM) {
    if(tid>=2){
      j=2;
			prime=1;
			while(j<tid){
				if(tid%j == 0){
					prime=0;
					arr[tid]=0;
					break;
				}
				j++;
			}
			if(prime == 1){
				arr[tid]=1;
			}
    }
		tid += blockDim.x * gridDim.x;
	}
}

int main(int argc, char* argv[]) {
	int i, j, *a, *d_a;
	double ms;

	a = new int[MAXIMUM + 1];

	hipMalloc( (void**) &d_a, MAXIMUM * sizeof(int) );

  printf("Starting...\n");
	ms = 0;
	for (i = 1; i <= N; i++) {
    hipMemcpy(d_a, a, MAXIMUM * sizeof(int), hipMemcpyHostToDevice);

		start_timer();

    for (j = 0; j <= MAXIMUM / 2; j++) {
      even<<<BLOCKS, THREADS>>>(d_a, MAXIMUM);
    }

		ms += stop_timer();
	}

  hipMemcpy(a, d_a, MAXIMUM * sizeof(int), hipMemcpyDeviceToHost);
  printf("Expanding the numbers that are prime to TOP_VALUE:\n");
  for (int i = 2; i < TOP_VALUE; i++) {
		if (a[i] == 1) {
			printf("%i ", i);
		}
	}
	//display_array("a", a);
  printf("avg time = %.5lf ms\n", (ms / N));

	hipFree(d_a);

	free(a);

  return 0;
}
